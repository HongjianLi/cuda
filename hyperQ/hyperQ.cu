
#include <hip/hip_runtime.h>
extern "C" __global__ void hyperQ(clock_t num_clocks)
{
	for (const clock_t threshold = clock() + num_clocks; clock() < threshold;);
}
