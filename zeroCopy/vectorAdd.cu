
#include <hip/hip_runtime.h>
extern "C" __global__ void vectorAdd(const float* const a, const float* const b, float* const c)
{
	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	c[gid] = a[gid] + b[gid];
}
