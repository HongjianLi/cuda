
#include <hip/hip_runtime.h>
extern "C" __global__ void spin(const clock_t num_clocks)
{
    const clock_t threshold = clock() + num_clocks;
    while (clock() < threshold);
}
