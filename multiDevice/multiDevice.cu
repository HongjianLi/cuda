
#include <hip/hip_runtime.h>
extern "C" __global__ void spin(clock_t num_clocks)
{
	for (const clock_t threshold = clock() + num_clocks; clock() < threshold;);
}
